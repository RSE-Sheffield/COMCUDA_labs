#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "hip/hip_runtime.h"
#include ""


#define A_WIDTH 1024
#define A_HEIGHT 1024
#define B_WIDTH 1024
#define B_HEIGHT 1024
#define C_WIDTH B_WIDTH
#define C_HEIGHT A_HEIGHT

#define NUM_SUBS (A_WIDTH / BLOCK_SIZE)

#define EPSILON 0.001f

__device__ float d_A[A_HEIGHT][A_WIDTH];
__device__ float d_B[B_HEIGHT][B_WIDTH];
__device__ float d_C[C_HEIGHT][C_WIDTH];

float h_A[A_HEIGHT][A_WIDTH];
float h_B[B_HEIGHT][B_WIDTH];
float h_C[C_HEIGHT][C_WIDTH];
float h_C_ref[C_HEIGHT][C_WIDTH];

void checkCUDAError(const char *msg);
void matrixMulCPU(float A[A_HEIGHT][A_WIDTH], float B[B_HEIGHT][B_WIDTH], float C[C_HEIGHT][C_WIDTH]);
int matrixMulTest(float C[C_HEIGHT][C_WIDTH], float Cref[C_HEIGHT][C_WIDTH]);
int requiredSM(int tpb);

__constant__ int BLOCK_SIZE;

__global__ void matrixMulCUDA()
{
	extern __shared__ float sm[];
	float *As = &sm[0];
	float *Bs = &sm[BLOCK_SIZE*BLOCK_SIZE];

	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	float Csub = 0;

	for (int i = 0; i < NUM_SUBS; i++){
		// Calculate indices of A and B matrix required to load the shared block of memory
		int a_x = (i*BLOCK_SIZE) + tx;
		int a_y = (by*BLOCK_SIZE) + ty;
		int b_x = (bx*BLOCK_SIZE) + tx;
		int b_y = (i*BLOCK_SIZE) + ty;

		As[(ty*BLOCK_SIZE) + tx] = d_A[a_y][a_x];
		Bs[(ty*BLOCK_SIZE) + tx] = d_B[b_y][b_x];

		// Sync to ensure sub matrix is fully loaded
		__syncthreads();

		// sum products of A and B sub matrices
		for (int k = 0; k < BLOCK_SIZE; k++)
		{
			Csub += As[(ty*BLOCK_SIZE) + k] * Bs[(k*BLOCK_SIZE) + tx];
		}

		// Sync to prevent run ahead (blocks loading new SM values before others have completed)
		__syncthreads();
	}

	// Store the product value of C matrix
	int c_x = (bx*BLOCK_SIZE) + tx;
	int c_y = (by*BLOCK_SIZE) + ty;
	d_C[c_y][c_x] = Csub;
}


int main(int argc, char **argv)
{
	unsigned int mem_size_A, mem_size_B, mem_size_C;
	unsigned int x, y, errors;
	int maxActiveBlocks, TPB, min_grid_size, block_size;
	float msec, occupancy;
	hipDeviceProp_t props;
	hipEvent_t start, stop;

	if (A_WIDTH != B_HEIGHT){
		printf("Error: A_HEIGHT and B_WIDTH do not match\n");
	}

	mem_size_A = sizeof(float)* A_WIDTH* A_HEIGHT;
	mem_size_B = sizeof(float)* B_WIDTH* B_HEIGHT;
	mem_size_C = sizeof(float)* C_WIDTH* C_HEIGHT;

	// Initialise A
	for (x = 0; x <A_WIDTH; x++)
	for (y = 0; y < A_HEIGHT; y++)
		h_A[x][y] = (float)rand() / RAND_MAX;
	// Initialise B
	for (x = 0; x <B_WIDTH; x++)
	for (y = 0; y < B_HEIGHT; y++)
		h_B[x][y] = (float)rand() / RAND_MAX;

	// copy host memory to device
	hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, mem_size_A);
	hipMemcpyToSymbol(HIP_SYMBOL(d_B), h_B, mem_size_B);
	checkCUDAError("CUDA memcpy");

	// Allocate CUDA events that we'll use for timing
	hipEventCreate(&start);
	hipEventCreate(&stop);
	checkCUDAError("CUDA event creation");

	// Calculate the block size
	hipOccupancyMaxPotentialBlockSizeVariableSMem(&min_grid_size, &TPB, matrixMulCUDA, requiredSM, 0);
	TPB = (int)pow(4, floor(log(TPB) / log(4))); //round to nearest square power 2
	block_size = (int) sqrt(TPB);
	hipMemcpyToSymbol(HIP_SYMBOL(BLOCK_SIZE), &block_size, sizeof(int));

	// calculate grid size and execute kernel
	dim3 threads(block_size, block_size);
	dim3 grid(C_WIDTH / block_size, C_HEIGHT / block_size);
	hipEventRecord(start);
	matrixMulCUDA << < grid, threads, requiredSM(block_size*block_size) >> >();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	checkCUDAError("CUDA kernel execution and timing");

	hipEventElapsedTime(&msec, start, stop);
	hipDeviceSynchronize();
	checkCUDAError("CUDA timing");

	// Compute the ocupancy
	hipGetDeviceProperties(&props, 0);
	hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, matrixMulCUDA, block_size*block_size, 0);
	occupancy = (maxActiveBlocks * block_size*block_size) / (float)(props.maxThreadsPerMultiProcessor);

	// Copy result from device to host
	hipMemcpyFromSymbol(h_C, HIP_SYMBOL(d_C), mem_size_C);
	checkCUDAError("CUDA memcpy results");

	// Compute reference CPU version
	matrixMulCPU(h_A, h_B, h_C_ref);

	// Check for errors
	errors = matrixMulTest(h_C, h_C_ref);
	if (errors)
		printf("%d total errors\n", errors);
	else
		printf("Test passed successfully\n");

	printf("Kernel time was %f with block size %d and theoretical occupancy of %f\n", msec, block_size, occupancy);

}

void matrixMulCPU(float A[A_HEIGHT][A_WIDTH], float B[C_HEIGHT][C_WIDTH], float C[C_HEIGHT][C_WIDTH])
{
	int x, y, k;
	for (y = 0; y < C_HEIGHT; y++){
		for (x = 0; x < C_WIDTH; x++){
			C[y][x] = 0;
			for (k = 0; k < A_WIDTH; k++){
				C[y][x] += A[y][k] * B[k][x];
			}
		}
	}

}

int matrixMulTest(float C[C_HEIGHT][C_WIDTH], float Cref[C_HEIGHT][C_WIDTH])
{
	int errors = 0;
	int y, x;

	for (y = 0; y < C_HEIGHT; y++){
		for (x = 0; x < C_WIDTH; x++){
			if (C[y][x] != Cref[y][x]){
				errors++;
				printf("Device item c[%d][%d] = %f does not mach host result %f\n", y, x, C[y][x], Cref[y][x]);
			}
		}
	}

	return errors;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

int requiredSM(int tpb){
	return (tpb*sizeof(float)* 2);
}
